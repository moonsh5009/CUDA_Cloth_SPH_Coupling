#include "hip/hip_runtime.h"
#include "CollisionDetection.cuh"
#include "CollisionResponse.cuh"
#include "CollisionSolver.h"

//-------------------------------------------------------------------------
void CollisionSolver::getSelfLastBvtts(
	BVHParam& clothBvh,
	Dvector<uint2>& lastBvtts, Dvector<uint>& LastBvttIds,
	uint lastBvhSize, uint& lastBvttSize)
{
	getNumLastBvtts_kernel << <divup(lastBvhSize, BLOCKSIZE), BLOCKSIZE >> > (
		clothBvh, LastBvttIds(), lastBvhSize);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::inclusive_scan(thrust::device_ptr<uint>(LastBvttIds.begin()), 
		thrust::device_ptr<uint>(LastBvttIds.begin() + lastBvhSize + 1u),
		thrust::device_ptr<uint>(LastBvttIds.begin()));
	CUDA_CHECK(hipMemcpy(&lastBvttSize, LastBvttIds() + lastBvhSize, sizeof(uint), hipMemcpyDeviceToHost));

	if (lastBvtts.size() < lastBvttSize)
		lastBvtts.resize(lastBvttSize);
	getLastBvtts_kernel << <divup(lastBvhSize, BLOCKSIZE), BLOCKSIZE >> > (
		clothBvh, lastBvtts(), LastBvttIds(), lastBvhSize);
	CUDA_CHECK(hipPeekAtLastError());
}
void CollisionSolver::getObstacleLastBvtts(
	BVHParam& clothBvh, BVHParam& obsBvh,
	Dvector<uint2>& lastBvtts, Dvector<uint>& LastBvttIds,
	uint lastBvhSize, uint& lastBvttSize)
{
	getNumLastBvtts_kernel << <divup(lastBvhSize, BLOCKSIZE), BLOCKSIZE >> > (
		clothBvh, obsBvh, LastBvttIds(), lastBvhSize);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::inclusive_scan(thrust::device_ptr<uint>(LastBvttIds.begin()),
		thrust::device_ptr<uint>(LastBvttIds.begin() + lastBvhSize + 1u),
		thrust::device_ptr<uint>(LastBvttIds.begin()));
	CUDA_CHECK(hipMemcpy(&lastBvttSize, LastBvttIds() + lastBvhSize, sizeof(uint), hipMemcpyDeviceToHost));

	if (lastBvtts.size() < lastBvttSize)
		lastBvtts.resize(lastBvttSize);
	getLastBvtts_kernel << <divup(lastBvhSize, BLOCKSIZE), BLOCKSIZE >> > (
		clothBvh, obsBvh, lastBvtts(), LastBvttIds(), lastBvhSize);
	CUDA_CHECK(hipPeekAtLastError());
}
//-------------------------------------------------------------------------
void CollisionSolver::getContactElements(
	ContactElems& ceParam, Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles)
{
	cloths->_bvh->refitProximity(cloths->d_fs(), cloths->d_ns(), cloths->d_nodePhases(), cloths->d_thicknesses());
	obstacles->_bvh->refitProximity(obstacles->d_fs(), obstacles->d_ns(), obstacles->d_nodePhases(), obstacles->d_thicknesses());

	ceParam._lastBvhSize = 1u << cloths->_bvh->_maxLevel - 1u;
	ceParam._lastBvttIds.resize(ceParam._lastBvhSize + 1u);

	ceParam._size = 0u;
	//ceParam.resize(0u);
	ceParam.d_tmp.memset(0);

	Dvector<uint2> selfLastBvtts;
	Dvector<uint2> obsLastBvtts;
	uint selfLastBvttSize, obsLastBvttSize;
	getSelfLastBvtts(
		cloths->_bvh->_param, selfLastBvtts, ceParam._lastBvttIds, ceParam._lastBvhSize, selfLastBvttSize);
	getObstacleLastBvtts(
		cloths->_bvh->_param, obstacles->_bvh->_param, obsLastBvtts, ceParam._lastBvttIds, ceParam._lastBvhSize, obsLastBvttSize);

	getNumSelfContactElements_LastBvtt_kernel << <divup(selfLastBvttSize, BLOCKSIZE), BLOCKSIZE, BLOCKSIZE * sizeof(uint) >> > (
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		selfLastBvtts(), selfLastBvttSize, ceParam.d_tmp());
	CUDA_CHECK(hipPeekAtLastError());
	getNumObstacleContactElements_LastBvtt_kernel << <divup(obsLastBvttSize, BLOCKSIZE), BLOCKSIZE, BLOCKSIZE * sizeof(uint) >> > (
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
		obsLastBvtts(), obsLastBvttSize, ceParam.d_tmp());
	CUDA_CHECK(hipPeekAtLastError());
	getNumSPHParticleContactElements_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param),
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		ceParam.d_tmp());
	CUDA_CHECK(hipPeekAtLastError());
	getNumSPHParticleContactElements_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param),
		*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
		ceParam.d_tmp());
	CUDA_CHECK(hipPeekAtLastError());

	CUDA_CHECK(hipMemcpy(&ceParam._size, ceParam.d_tmp(), sizeof(uint), hipMemcpyDeviceToHost));
	if (ceParam._size > 0u) {
		if (ceParam._elems.size() < ceParam._size)
			ceParam.resize();
		ceParam.d_tmp.memset(0);
		getSelfContactElements_LastBvtt_kernel << <divup(selfLastBvttSize, BLOCKSIZE), BLOCKSIZE >> > (
			*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
			selfLastBvtts(), selfLastBvttSize, ceParam.param());
		CUDA_CHECK(hipPeekAtLastError());
		getObstacleContactElements_LastBvtt_kernel << <divup(obsLastBvttSize, BLOCKSIZE), BLOCKSIZE >> > (
			*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
			*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
			obsLastBvtts(), obsLastBvttSize, ceParam.param());
		CUDA_CHECK(hipPeekAtLastError());
		getSPHParticleContactElements_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
			*((SPHParticleParam*)sphParticles->_param),
			*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
			ceParam.param());
		CUDA_CHECK(hipPeekAtLastError());
		getSPHParticleContactElements_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
			*((SPHParticleParam*)sphParticles->_param),
			*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
			ceParam.param());
		CUDA_CHECK(hipPeekAtLastError());
		CUDA_CHECK(hipMemcpy(&ceParam._size, ceParam.d_tmp(), sizeof(uint), hipMemcpyDeviceToHost));
	}
	CUDA_CHECK(hipDeviceSynchronize());
	printf("Collision %d\n", ceParam._size);
}
void CollisionSolver::getClothCCDtime(
	Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles,
	const REAL dt, REAL* minTime)
{
#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
#endif
	* minTime = 10.0;

	REAL* d_minTime;
	CUDA_CHECK(hipMalloc((void**)&d_minTime, sizeof(REAL)));
	CUDA_CHECK(hipMemcpy(d_minTime, minTime, sizeof(REAL), hipMemcpyHostToDevice));

	cloths->_bvh->refitCCD(cloths->d_fs(), cloths->d_ns(), cloths->d_vs(), cloths->d_nodePhases(), cloths->d_thicknesses(), dt);
	obstacles->_bvh->refitCCD(obstacles->d_fs(), obstacles->d_ns(), obstacles->d_vs(), obstacles->d_nodePhases(), obstacles->d_thicknesses(), dt);

	getSelfCCDtime_kernel << <divup(cloths->_numFaces, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		dt, d_minTime);
	CUDA_CHECK(hipPeekAtLastError());
	getObstacleCCDtime_kernel << <divup(cloths->_numFaces, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
		dt, d_minTime);
	CUDA_CHECK(hipPeekAtLastError());
	getSPHParticleCCDtime_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param),
		*cloths->_param, cloths->_bvh->_param, cloths->_RTri->_param,
		dt, d_minTime);
	CUDA_CHECK(hipPeekAtLastError());
	getSPHParticleCCDtime_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param),
		*obstacles->_param, obstacles->_bvh->_param, obstacles->_RTri->_param,
		dt, d_minTime);
	CUDA_CHECK(hipPeekAtLastError());

	CUDA_CHECK(hipMemcpy(minTime, d_minTime, sizeof(REAL), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_minTime));

#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	printf("CollisionSolver::getClothCCDtime: %lf msec\n", (CNOW - timer) / 10000.0);
#endif
}
//-------------------------------------------------------------------------
void CollisionSolver::MakeRigidImpactZone(
	const ContactElems& d_ceParam,
	RIZone& h_riz, DRIZone& d_riz,
	const PrefixArray<uint>& clothNbNs, const PrefixArray<uint>& obsNbNs)
{
	vector<ContactElem> h_ceParam(d_ceParam._size);
	CUDA_CHECK(hipMemcpy(&h_ceParam[0], d_ceParam._elems(), d_ceParam._size * sizeof(ContactElem), hipMemcpyDeviceToHost));

	uint2 nodes[4];
	for (uint ice = 0; ice < h_ceParam.size(); ice++) {
		if (!h_ceParam[ice]._isCCD)
			continue;

		nodes[0].x = h_ceParam[ice]._i[0];
		nodes[1].x = h_ceParam[ice]._i[1];
		nodes[2].x = h_ceParam[ice]._i[2];
		nodes[3].x = h_ceParam[ice]._i[3];
		nodes[0].y = h_ceParam[ice]._type[0];
		nodes[1].y = h_ceParam[ice]._type[1];
		nodes[2].y = h_ceParam[ice]._type[2];
		nodes[3].y = h_ceParam[ice]._type[3];

		set<uint> ind_inc;
		for (uint i = 0; i < 4; i++) {
			uint2 ino = nodes[i];
			for (uint iriz = 0; iriz < h_riz.size(); iriz++) {
				if (h_riz[iriz].find(ino) != h_riz[iriz].end())
					ind_inc.insert(iriz);
				else {
					if (ino.y == TYPE_MESH_CLOTH) {
						for (uint j = clothNbNs._index[ino.x]; j < clothNbNs._index[ino.x + 1u]; j++) {
							uint2 jno = make_uint2(clothNbNs._array[j], 0u);
							if (h_riz[iriz].find(jno) != h_riz[iriz].end()) {
								ind_inc.insert(iriz);
								break;
							}
						}
					}
					else if (ino.y == TYPE_MESH_OBSTACLE) {
						for (uint j = obsNbNs._index[ino.x]; j < obsNbNs._index[ino.x + 1u]; j++) {
							uint2 jno = make_uint2(obsNbNs._array[j], 1u);
							if (h_riz[iriz].find(jno) != h_riz[iriz].end()) {
								ind_inc.insert(iriz);
								break;
							}
						}
					}
				}
			}
		}
		uint ind0;
		if (ind_inc.size() == 0) {
			ind0 = (uint)h_riz.size();
			h_riz.resize(ind0 + 1u);
		}
		else if (ind_inc.size() == 1u)
			ind0 = *(ind_inc.begin());
		else {
			RIZone h_riz1;
			for (uint iriz = 0; iriz < h_riz.size(); iriz++) {
				if (ind_inc.find(iriz) != ind_inc.end()) continue;
				h_riz1.push_back(h_riz[iriz]);
			}
			ind0 = (uint)h_riz1.size();
			h_riz1.resize(ind0 + 1);
			for (auto itr = ind_inc.begin(); itr != ind_inc.end(); itr++) {
				uint ind1 = *itr;
				for (auto jtr = h_riz[ind1].begin(); jtr != h_riz[ind1].end(); jtr++)
					h_riz1[ind0].insert(*jtr);
			}
			h_riz = h_riz1;
		}

		for (uint i = 0; i < 4; i++)
			h_riz[ind0].insert(nodes[i]);
	}

	vector<uint2> h_ids;
	vector<uint> h_zones;
	h_zones.resize(h_riz.size() + 1u);
	h_zones[0] = 0;
	for (uint i = 0; i < h_riz.size(); i++)
		h_zones[i + 1] = h_zones[i] + h_riz[i].size();

	h_ids.resize(h_zones.back());
	uint n = 0;
	for (uint i = 0; i < h_riz.size(); i++) {
		for (auto jtr = h_riz[i].begin(); jtr != h_riz[i].end(); jtr++)
			h_ids[n++] = *jtr;
	}

	d_riz._ids = h_ids;
	d_riz._zones = h_zones;
}
bool CollisionSolver::ResolveRigidImpactZone(
	ContactElems& ceParam,
	RIZone& h_riz, DRIZone& d_riz,
	const ObjParam& clothParam, const ObjParam& obsParam, const SPHParticleParam& sphParticles,
	const PrefixArray<uint>& clothNbNs, const PrefixArray<uint>& obsNbNs,
	const REAL dt)
{
#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
#endif
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
	bool result = false;
	if (ceParam._size) {
		bool* d_applied;

		CUDA_CHECK(hipMalloc((void**)&d_applied, sizeof(bool)));
		CUDA_CHECK(hipMemset(d_applied, 0, sizeof(bool)));

		compDetectedRigidImpactZone_CE_kernel << <divup(ceParam._size, BLOCKSIZE), BLOCKSIZE >> > (
			ceParam.param(), clothParam, obsParam, sphParticles, dt, d_applied);
		CUDA_CHECK(hipPeekAtLastError());

		CUDA_CHECK(hipMemcpy(&result, d_applied, sizeof(bool), hipMemcpyDeviceToHost));
		CUDA_CHECK(hipFree(d_applied));

		if (result) {
			MakeRigidImpactZone(ceParam, h_riz, d_riz, clothNbNs, obsNbNs);
			ApplyRigidImpactZone_kernel << <divup(d_riz._zones.size() - 1u, BLOCKSIZE), BLOCKSIZE >> > (
				clothParam, obsParam, sphParticles, d_riz.param(), dt);
			CUDA_CHECK(hipPeekAtLastError());
		}
	}
	CUDA_CHECK(hipDeviceSynchronize());
	printf("Resolve Cloth Collision RIZ: %lf msec\n", (CNOW - timer) / 10000.0);

	return result;

#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	printf("Resolve Cloth Collision RIZ: %lf msec\n", (CNOW - timer) / 10000.0);
#endif
}
void CollisionSolver::compRigidImpactZone(
	ContactElems& ceParam,
	RIZone& h_riz, DRIZone& d_riz,
	Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles,
	const REAL dt) 
{
#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
#endif

	uint itr = 0u;
	//Dvector<uint> clothZones(clothParam._numNodes);
	//Dvector<uint> obsZones(obsParam._numNodes);
	while (ResolveRigidImpactZone(
		ceParam, h_riz, d_riz, *cloths->_param, *obstacles->_param, *((SPHParticleParam*)sphParticles->_param),
		cloths->h_nbNs, obstacles->h_nbNs, dt))
		itr++;
	//-------------------------------------------------------------------------------
	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("step 3: %lf msec\n", (CNOW - timer) / 10000.0);
	//timer = CNOW;
	//-------------------------------------------------------------------------------
	if (itr > 0u) printf("Rigid Impact Zone %d\n", itr);

#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	printf("CollisionSolver::compClothRigidImpactZone: %lf msec\n", (CNOW - timer) / 10000.0);
#endif
}
//-------------------------------------------------------------------------
void CollisionSolver::compCollisionImpulse(
	ContactElems& ceParam,
	Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles,
	bool isProximity, const REAL dt)
{
#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
#endif

	compCollisionImpulse_CE_kernel << <divup(ceParam._size, BLOCKSIZE), BLOCKSIZE >> > (
		ceParam.param(), *cloths->_param, *obstacles->_param, *((SPHParticleParam*)sphParticles->_param), isProximity, dt);
	CUDA_CHECK(hipPeekAtLastError());

#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	printf("CollisionSolver::compClothCollisionImpulse: %lf msec\n", (CNOW - timer) / 10000.0);
#endif
}
//-------------------------------------------------------------------------
bool CollisionSolver::applyImpulse(
	Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles, REAL dt)
{
#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;
#endif

	bool result = false;
	bool* d_applied;

	CUDA_CHECK(hipMalloc((void**)&d_applied, sizeof(bool)));
	CUDA_CHECK(hipMemset(d_applied, 0, sizeof(bool)));

	applyClothCollision_kernel << <divup(cloths->_numNodes, BLOCKSIZE), BLOCKSIZE >> > (
		*cloths->_param, dt, d_applied);
	CUDA_CHECK(hipPeekAtLastError());

	applySPHParticleCollision_kernel << <divup(sphParticles->_numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param), dt, d_applied);
	CUDA_CHECK(hipPeekAtLastError());

	CUDA_CHECK(hipMemcpy(&result, d_applied, sizeof(bool), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_applied));
	if (result) {
		/*Dvector<REAL> buffer;
		buffer = cloths->d_impulses;
		for (int s = 0; s < COLLISION_SMOOTHING; s++) {
			SmoothingImpulse_kernel << <divup(cloths->_numNodes, BLOCKSIZE), BLOCKSIZE >> > (
				*cloths->_param, cloths->d_nbNs._index(), cloths->d_nbNs._array(), cloths->d_impulses(), buffer());
			CUDA_CHECK(hipPeekAtLastError());
			cloths->d_impulses = buffer;
		}*/
		applyClothImpulse_kernel << <divup(cloths->_numNodes, BLOCKSIZE), BLOCKSIZE >> > (
			*cloths->_param);
		CUDA_CHECK(hipPeekAtLastError());
	}

#ifdef COLLISION_TESTTIMER
	CUDA_CHECK(hipDeviceSynchronize());
	printf("CollisionSolver::applyImpulse: %lf msec\n", (CNOW - timer) / 10000.0);
#endif

	return result;
}
void CollisionSolver::compCollisionIteration(
	ContactElems& ceParam, Cloth* cloths, Obstacle* obstacles, SPHParticle* sphParticles, const REAL dt)
{
	bool isDetected;
	REAL subDt = dt;
	REAL minTime;
	uint itr;

	getContactElements(ceParam, cloths, obstacles, sphParticles);
	for (itr = 0u; itr < 100u; itr++) {
		cloths->d_impulses.memset(0);
		obstacles->d_impulses.memset(0);
		sphParticles->d_impulses.memset(0);
		cloths->d_colWs.memset(0);
		obstacles->d_colWs.memset(0);
		sphParticles->d_colWs.memset(0);
		compCollisionImpulse(ceParam, cloths, obstacles, sphParticles, true, subDt);
		isDetected = applyImpulse(cloths, obstacles, sphParticles, subDt);
		if (!isDetected)
			break;
	}

	getClothCCDtime(cloths, obstacles, sphParticles, subDt, &minTime);

	while (minTime <= 1.0) {
		printf("CCD %.20f\n", minTime);
		minTime *= 0.8;
		Simulation::updatePosition(cloths->d_ns, cloths->d_vs, subDt * minTime);
		Simulation::updatePosition(obstacles->d_ns, obstacles->d_vs, subDt * minTime);
		Simulation::updatePosition(sphParticles->d_xs, sphParticles->d_vs, subDt * minTime);
		subDt -= subDt * minTime;

		getContactElements(ceParam, cloths, obstacles, sphParticles);
		for (itr = 0u; itr < 100u; itr++) {
			cloths->d_impulses.memset(0);
			obstacles->d_impulses.memset(0);
			sphParticles->d_impulses.memset(0);
			cloths->d_colWs.memset(0);
			obstacles->d_colWs.memset(0);
			sphParticles->d_colWs.memset(0);
			compCollisionImpulse(ceParam, cloths, obstacles, sphParticles, false, subDt);
			isDetected = applyImpulse(cloths, obstacles, sphParticles, subDt);
			if (!isDetected)
				break;
		}

		if (/*(minTime < 0.0001 || subDt < 0.1 * dt) && */isDetected) {
			RIZone h_riz;
			DRIZone d_riz;
			compRigidImpactZone(ceParam, h_riz, d_riz, cloths, obstacles, sphParticles, subDt);
		}

		getClothCCDtime(cloths, obstacles, sphParticles, subDt, &minTime);
	}
	Simulation::updatePosition(cloths->d_ns, cloths->d_vs, subDt);
	Simulation::updatePosition(sphParticles->d_xs, sphParticles->d_vs, subDt);
	Simulation::updateVelocity(cloths->d_n0s, cloths->d_ns, cloths->d_vs, 1.0 / dt);
	Simulation::updateVelocity(sphParticles->d_x0s, sphParticles->d_xs, sphParticles->d_vs, 1.0 / dt);
	cloths->d_ns = cloths->d_n0s;
	obstacles->d_ns = obstacles->d_n0s;
	sphParticles->d_xs = sphParticles->d_x0s;
}