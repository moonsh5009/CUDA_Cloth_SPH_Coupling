#include "hip/hip_runtime.h"
#include "SpatialHashing.cuh"

void SpatialHashing::sort(SPHParticle* particles) {
#ifdef HASH_TIMER
	hipDeviceSynchronize();
	ctimer timer = CNOW;
#endif

	uint numParticles = particles->_numParticles;
	if (!numParticles)
		return;

	initParticle(numParticles);

	Dvector<REAL> oldXs(numParticles * 3u);
	Dvector<REAL> oldVs(numParticles * 3u);
	Dvector<REAL> oldSs(numParticles);
	Dvector<REAL> oldRelaxTs(numParticles);
	Dvector<uint> oldPhase(numParticles);

	CUDA_CHECK(hipMemcpy(oldXs(), particles->d_xs(), numParticles * sizeof(REAL3), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(oldVs(), particles->d_vs(), numParticles * sizeof(REAL3), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(oldSs(), particles->d_ss(), numParticles * sizeof(REAL), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(oldRelaxTs(), particles->d_relaxTs(), numParticles * sizeof(REAL), hipMemcpyDeviceToDevice));
	CUDA_CHECK(hipMemcpy(oldPhase(), particles->d_phases(), numParticles * sizeof(uint), hipMemcpyDeviceToDevice));

	initHashZindex_kernel << <divup(numParticles, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> >
		((REAL3*)particles->d_xs(), _param);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(thrust::device_ptr<uint>((uint*)_keys()),
		thrust::device_ptr<uint>(((uint*)_keys()) + _numParticles),
		thrust::device_ptr<uint2>((uint2*)_ids()));

	ParticleZSort_kernel << <divup(numParticles, BLOCKSIZE), BLOCKSIZE >> >
		(*((SPHParticleParam*)particles->_param), oldXs(), oldVs(), oldSs(), oldRelaxTs(), oldPhase(), _ids());
	CUDA_CHECK(hipPeekAtLastError());

	particles->d_xs.copyToHost(particles->h_xs);
	particles->d_vs.copyToHost(particles->h_vs);
	particles->d_ss.copyToHost(particles->h_ss);
	particles->d_relaxTs.copyToHost(particles->h_relaxTs);
	particles->d_phases.copyToHost(particles->h_phases);

#ifdef HASH_TIMER
	hipDeviceSynchronize();
	printf("SpatialHashing::sort: %f msec\n", (CNOW - timer) / 10000.0);
#endif
}
void SpatialHashing::insert(SPHParticle* sphParticles, PoreParticle* poreParticles, BoundaryParticle* boundaryParticles) {
#ifdef HASH_TIMER
	hipDeviceSynchronize();
	ctimer timer = CNOW;
#endif

	uint numParticles = sphParticles->_numParticles + poreParticles->_numParticles + boundaryParticles->_numParticles;
	if (!numParticles)
		return;

	initParticle(numParticles);
	
	initHash_kernel << <divup(numParticles, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> >
		(*sphParticles->_param, *poreParticles->_param, *boundaryParticles->_param, _param);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(thrust::device_ptr<uint>((uint*)_keys()),
		thrust::device_ptr<uint>(((uint*)_keys()) + _numParticles),
		thrust::device_ptr<uint2>((uint2*)_ids()));

	reorderHash_kernel << <divup(numParticles, MAX_BLOCKSIZE), MAX_BLOCKSIZE, (MAX_BLOCKSIZE + 1) * sizeof(uint) >> >
		(_param);
	CUDA_CHECK(hipPeekAtLastError());

#ifdef HASH_TIMER
	hipDeviceSynchronize();
	printf("SpatialHashing::insert: %f msec\n", (CNOW - timer) / 10000.0);
#endif
}
void SpatialHashing::getNeighbors(SPHParticle* sphParticles, PoreParticle* poreParticles, BoundaryParticle* boundaryParticles) {
#ifdef HASH_TIMER
	hipDeviceSynchronize();
	ctimer timer = CNOW;
#endif
	uint numParticles = sphParticles->_numParticles + poreParticles->_numParticles + boundaryParticles->_numParticles;
	if (!numParticles)
		return;

	compNumNeighbors_kernel << <divup(numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param), 
		*((PoreParticleParam*)poreParticles->_param),
		*((BoundaryParticleParam*)boundaryParticles->_param), _param);
	CUDA_CHECK(hipPeekAtLastError());

	uint numSPHNeis = 0u;
	uint numPoreNeis = 0u;
	uint numBoundaryNeis = 0u;

	if (sphParticles->_numParticles) {
		thrust::inclusive_scan(thrust::device_ptr<uint>(sphParticles->d_ineis.begin()),
			thrust::device_ptr<uint>(sphParticles->d_ineis.end()), sphParticles->d_ineis.begin());
		CUDA_CHECK(hipMemcpy(&numSPHNeis, sphParticles->d_ineis() + sphParticles->_numParticles, sizeof(uint), hipMemcpyDeviceToHost));
	}
	if (poreParticles->_numParticles) {
		thrust::inclusive_scan(thrust::device_ptr<uint>(poreParticles->d_ineis.begin()),
			thrust::device_ptr<uint>(poreParticles->d_ineis.end()), poreParticles->d_ineis.begin());
		CUDA_CHECK(hipMemcpy(&numPoreNeis, poreParticles->d_ineis() + poreParticles->_numParticles, sizeof(uint), hipMemcpyDeviceToHost));
	}
	if (boundaryParticles->_numParticles) {
		thrust::inclusive_scan(thrust::device_ptr<uint>(boundaryParticles->d_ineis.begin()),
			thrust::device_ptr<uint>(boundaryParticles->d_ineis.end()), boundaryParticles->d_ineis.begin());
		CUDA_CHECK(hipMemcpy(&numBoundaryNeis, boundaryParticles->d_ineis() + boundaryParticles->_numParticles, sizeof(uint), hipMemcpyDeviceToHost));
	}

	sphParticles->d_neis.resize(numSPHNeis);
	poreParticles->d_neis.resize(numPoreNeis);
	boundaryParticles->d_neis.resize(numBoundaryNeis);
	sphParticles->_param->_neis = sphParticles->d_neis._list;
	poreParticles->_param->_neis = poreParticles->d_neis._list;
	boundaryParticles->_param->_neis = boundaryParticles->d_neis._list;

	getNeighbors_kernel << <divup(numParticles, BLOCKSIZE), BLOCKSIZE >> > (
		*((SPHParticleParam*)sphParticles->_param),
		*((PoreParticleParam*)poreParticles->_param),
		*((BoundaryParticleParam*)boundaryParticles->_param), _param);
	CUDA_CHECK(hipPeekAtLastError());

#ifdef HASH_TIMER
	hipDeviceSynchronize();
	printf("SpatialHashing::getNeighbors: %f msec\n", (CNOW - timer) / 10000.0);
#endif
}